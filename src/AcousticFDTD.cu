#include "hip/hip_runtime.h"
#include "AcousticFDTD.h"

#define _USE_MATH_DEFINES
#include <math.h>

#include <CudaHelper.h>

AcousticFDTD::AcousticFDTD(glm::ivec2 & gridSize, GLuint * vbo, unsigned int blockSize, unsigned int dataPerThread)
{
	_gridSize = gridSize;
	if(dataPerThread > 0 && dataPerThread <= gridSize.x)
	{
		_dataPerThread = glm::ivec2(dataPerThread, dataPerThread);
	}
	else
	{
		_dataPerThread = glm::ivec2(1, 1);
	}

	if(blockSize > 0 && blockSize <= 32)
	{
		_cudaBlockSize = dim3(blockSize, blockSize);
	}
	else
	{
		_cudaBlockSize = dim3(32, 32);
	}
	const int bx = (gridSize.x + _cudaBlockSize.x - 1) / _cudaBlockSize.x;
	const int by = (gridSize.y + _cudaBlockSize.y - 1) / _cudaBlockSize.y;
	_cudaGridSize = dim3(bx, by);

	CudaSafeCall(hipGraphicsGLRegisterBuffer(&_cudaVboRes, *vbo, cudaGraphicsMapFlagsNone));
	
	for(unsigned int i = 0; i < 2; ++i)
	{
		CudaSafeCall(hipMalloc((void **)&_grid[i], (gridSize.x + 1) * (gridSize.y + 1) * sizeof(SpacePoint)));
		CudaSafeCall(hipMemset(_grid[i], 0, (gridSize.x + 1) * (gridSize.y + 1) * sizeof(SpacePoint)));
		CudaSafeCall(hipMalloc((void **)&_murX[i], gridSize.y * 4 * sizeof(float)));
		CudaSafeCall(hipMemset(_murX[i], 0, gridSize.y * 4 * sizeof(float)));
		CudaSafeCall(hipMalloc((void **)&_murY[i], gridSize.x * 4 * sizeof(float)));
		CudaSafeCall(hipMemset(_murY[i], 0, gridSize.x * 4 * sizeof(float)));
	}
}

AcousticFDTD::~AcousticFDTD()
{
	CudaSafeCall(hipGraphicsUnregisterResource(_cudaVboRes));

	CudaSafeCall(hipFree(_grid[0]));
	CudaSafeCall(hipFree(_grid[1]));
	CudaSafeCall(hipFree(_murX[0]));
	CudaSafeCall(hipFree(_murX[1]));
	CudaSafeCall(hipFree(_murY[0]));
	CudaSafeCall(hipFree(_murY[1]));
}

__global__ void updateV(glm::ivec2 dataPerThread, glm::ivec2 gridSize, AcousticFDTD::SpacePoint * inGrid,
		AcousticFDTD::SpacePoint * outGrid, float dtOverDx, float density)
{
	const int startI = (blockIdx.x * blockDim.x + threadIdx.x) * dataPerThread.x;
	const int startJ = (blockIdx.y * blockDim.y + threadIdx.y) * dataPerThread.y;

	unsigned int rangeI = (startI + dataPerThread.x) < gridSize.x ? (startI + dataPerThread.x) : gridSize.x;
	unsigned int rangeJ = (startJ + dataPerThread.y) < gridSize.y ? (startJ + dataPerThread.y) : gridSize.y;

	for(unsigned int i = startI == 0 ? 1 : startI; i < rangeI; ++i)
	{
		for(unsigned int j = startJ; j < rangeJ; ++j)
		{
			outGrid[((i * gridSize.y) + j)].vx += -dtOverDx / density * (inGrid[(i * gridSize.y) + j].soundPressure - inGrid[((i - 1) * gridSize.y) + j].soundPressure);
		}
	}

	for(unsigned int i = startI; i < rangeI; ++i)
	{
		for(unsigned int j = startJ == 0 ? 1 : startJ; j < rangeJ; ++j)
		{
			outGrid[((i * gridSize.y) + j)].vy += -dtOverDx / density * (inGrid[(i * gridSize.y) + j].soundPressure - inGrid[i * gridSize.y + (j - 1)].soundPressure);
		}
	}
}

__global__ void updateP(glm::ivec2 dataPerThread, glm::ivec2 gridSize, AcousticFDTD::SpacePoint * inGrid,
		AcousticFDTD::SpacePoint * outGrid, float dtOverDx, float bulkModulus)
{
	const int startI = (blockIdx.x * blockDim.x + threadIdx.x) * dataPerThread.x;
	const int startJ = (blockIdx.y * blockDim.y + threadIdx.y) * dataPerThread.y;

	unsigned int rangeI = (startI + dataPerThread.x) < gridSize.x ? (startI + dataPerThread.x) : gridSize.x;
	unsigned int rangeJ = (startJ + dataPerThread.y) < gridSize.y ? (startJ + dataPerThread.y) : gridSize.y;

	for(unsigned int i = startI; i < rangeI; ++i)
	{
		for(unsigned int j = startJ; j < rangeJ; ++j)
		{
			outGrid[((i * gridSize.y) + j)].soundPressure += -(bulkModulus * dtOverDx) * ((inGrid[((i + 1) * gridSize.y) + j].vx - inGrid[(i * gridSize.y) + j].vx) + (inGrid[(i * gridSize.y) + (j + 1)].vy - inGrid[(i * gridSize.y) + j].vy));
		}
	}
}

__global__ void mur2nd(glm::ivec2 dataPerThread, glm::ivec2 gridSize, AcousticFDTD::SpacePoint * inGrid, AcousticFDTD::SpacePoint * outGrid,
		float * murX1, float * murX2, float * murY1, float * murY2, float dt, float dx, float density, float bulkModulus)
{
	const int startI = (blockIdx.x * blockDim.x + threadIdx.x) * dataPerThread.x;
	const int startJ = (blockIdx.y * blockDim.y + threadIdx.y) * dataPerThread.y;

	unsigned int rangeI = (startI + dataPerThread.x) < (gridSize.x - 2) ? (startI + dataPerThread.x) : (gridSize.x - 2);
	unsigned int rangeJ = (startJ + dataPerThread.y) < (gridSize.y - 2) ? (startJ + dataPerThread.y) : (gridSize.y - 2);

	float v = sqrt(bulkModulus/density);	// Wave velocity
	int i,j;

	for(i = startI < 2 ? 2 : startI; i < rangeI; ++i){
		outGrid[(i * gridSize.y)].soundPressure = - murY2[(i * 4 + 1)]
		                                         + (v*dt-dx)/(v*dt+dx) * ( inGrid[(i * gridSize.y + 1)].soundPressure + murY2[(i * 4)] )
		                                         + (2.0*dx)/(v*dt+dx) * ( murY1[i * 4] + murY1[i * 4 + 1] )
		                                         + (dx*v*v*dt*dt)/(2.0*dx*dx*(v*dt+dx))
		                                           * ( murY1[(i+1) * 4] - 2.0 * murY1[i * 4]
		                                               + murY1[(i-1) * 4] + murY1[(i+1) * 4 + 1]
		                                               - 2.0 * murY1[i* 4 + 1] + murY1[(i-1) * 4 + 1] );
		outGrid[(i * gridSize.y) + (gridSize.y - 1)].soundPressure = - murY2[(i * 4) + 2]
		                                                             + (v*dt-dx)/(v*dt+dx) * ( inGrid[(i * gridSize.y) + gridSize.y-2].soundPressure + murY2[i * 4 + 3] )
		                                                             + (2.0*dx)/(v*dt+dx) * ( murY1[i * 4 + 3] + murY1[i * 4 + 2] )
		                                                             + (dx*v*v*dt*dt)/(2.0*dx*dx*(v*dt+dx))
		                                                               * ( murY1[(i+1) * 4 + 3] - 2.0 * murY1[i * 4 + 3]
		                                                                   + murY1[(i-1) * 4 + 3] + murY1[(i+1) * 4 + 2]
		                                                                   - 2.0 * murY1[i * 4 + 2] + murY1[(i-1) * 4 + 2] );
	}
	for(j = startJ < 2 ? 2 : startJ; j < rangeJ; ++j){
		outGrid[j].soundPressure = - murX2[1 + j * 4]
		                         + (v*dt-dx)/(v*dt+dx) * ( inGrid[1 * gridSize.y + j].soundPressure + murX2[0 + j * 4] )
		                         + (2.0*dx)/(v*dt+dx) * ( murX1[0 + j * 4] + murX1[1 + j * 4] )
		                         + (dx*v*v*dt*dt)/(2.0*dx*dx*(v*dt+dx))
		                           * ( murX1[0 + (j+1) * 4] - 2.0 * murX1[0 + j * 4]
		                               + murX1[0 + (j-1) * 4] + murX1[1 + (j+1) * 4]
		                               - 2.0 * murX1[1 + j * 4] + murX1[1 + (j-1) * 4] );
		outGrid[(gridSize.x-1) * gridSize.y + j].soundPressure = - murX2[2 + j * 4]
		                                                         + (v*dt-dx)/(v*dt+dx) * ( inGrid[(gridSize.x-2) * gridSize.y + j].soundPressure + murX2[3 + j * 4] )
		                                                         + (2.0*dx)/(v*dt+dx) * ( murX1[3 + j * 4] + murX1[2 + j * 4] )
		                                                         + (dx*v*v*dt*dt)/(2.0*dx*dx*(v*dt+dx))
		                                                           * ( murX1[3 + (j+1) * 4] - 2.0 * murX1[3 + j * 4]
		                                                               + murX1[3 + (j-1) * 4] + murX1[2 + (j+1) * 4]
		                                                               - 2.0 * murX1[2 + j * 4] + murX1[2 + (j-1) * 4] );
	}

	// Mur's 1st Order Absorption for 4 corners
	// computed by first thread
	if(startI == 0)
	{
		i = 1;
		outGrid[i * gridSize.y].soundPressure = murY1[i * 4 + 1] + (v*dt-dx)/(v*dt+dx) * (inGrid[i * gridSize.y + 1].soundPressure - murY1[i * 4]);
		outGrid[i * gridSize.y + gridSize.y-1].soundPressure = murY1[i * 4 + 2] + (v*dt-dx)/(v*dt+dx) * (inGrid[i * gridSize.y + gridSize.y-2].soundPressure - murY1[i * 4 + 3]);
		i = gridSize.x-2;
		outGrid[i * gridSize.y].soundPressure = murY1[i * 4 + 1] + (v*dt-dx)/(v*dt+dx) * (inGrid[i* gridSize.y + 1].soundPressure - murY1[i * 4]);
		outGrid[i * gridSize.y + gridSize.y-1].soundPressure = murY1[i * 4 + 2] + (v*dt-dx)/(v*dt+dx) * (inGrid[i * gridSize.y + gridSize.y-2].soundPressure - murY1[i * 4 + 3]);
	}
	if(startJ == 0)
	{
		j = 1;
		outGrid[0 * gridSize.y + j].soundPressure = murX1[1 + j * 4] + (v*dt-dx)/(v*dt+dx) * (inGrid[1 * gridSize.y + j].soundPressure - murX1[0 + j * 4]);
		outGrid[(gridSize.x-1) * gridSize.y + j].soundPressure = murX1[2 + j * 4] + (v*dt-dx)/(v*dt+dx) * (inGrid[(gridSize.x-2) * gridSize.y + j].soundPressure - murX1[3 + j * 4]);
		j = gridSize.y - 2;
		outGrid[0 * gridSize.y + j].soundPressure = murX1[1 + j * 4] + (v*dt-dx)/(v*dt+dx) * (inGrid[1 * gridSize.y + j].soundPressure - murX1[0 + j * 4]);
		outGrid[(gridSize.x-1) * gridSize.y + j].soundPressure = murX1[2+ j * 4] + (v*dt-dx)/(v*dt+dx) * (inGrid[(gridSize.x-2) * gridSize.y + j].soundPressure - murX1[3 + j * 4]);
	}
}

__global__ void mur2ndCopy(glm::ivec2 dataPerThread, glm::ivec2 gridSize, AcousticFDTD::SpacePoint * grid,
		float * murX1, float * murX2, float * murY1, float * murY2)
{
	const int startI = (blockIdx.x * blockDim.x + threadIdx.x) * dataPerThread.x;
	const int startJ = (blockIdx.y * blockDim.y + threadIdx.y) * dataPerThread.y;

	unsigned int rangeI = (startI + dataPerThread.x) < gridSize.x ? (startI + dataPerThread.x) : gridSize.x;
	unsigned int rangeJ = (startJ + dataPerThread.y) < gridSize.y ? (startJ + dataPerThread.y) : gridSize.y;

	for(unsigned int i = startI; i < rangeI; ++i)
	{
		// Copy 1st Old Values to 2nd Old Values
		for(unsigned int j = 0; j < 4; ++j)
		{
			murY2[i * 4 + j] = murY1[i * 4 + j];
		}

		// Copy Present Values
		murY1[i * 4 + 0] = grid[i * gridSize.y + 0].soundPressure;
		murY1[i * 4 + 1] = grid[i * gridSize.y + 1].soundPressure;
		murY1[i * 4 + 2] = grid[i * gridSize.y + gridSize.y-2].soundPressure;
		murY1[i * 4 + 3] = grid[i * gridSize.y + gridSize.y-1].soundPressure;
	}
	for(unsigned int j = startJ; j < rangeJ; ++j){
		// Copy 1st Old Values to 2nd Old Values
		for(unsigned int i = 0; i < 4; ++i)
		{
			murX2[i + j * 4] = murX1[i + j * 4];
		}

		// Copy Present Values
		murX1[0 + j * 4] = grid[0 * gridSize.y + j].soundPressure;
		murX1[1 + j * 4] = grid[1 * gridSize.y + j].soundPressure;
		murX1[2 + j * 4] = grid[(gridSize.x-2) * gridSize.y + j].soundPressure;
		murX1[3 + j * 4] = grid[(gridSize.x-1) * gridSize.y + j].soundPressure;
	}
}

__global__ void updateColors(glm::ivec2 dataPerThread, glm::ivec2 gridSize, AcousticFDTD::SpacePoint * grid, Vertex * vertexPointer)
{
	const int startI = (blockIdx.x * blockDim.x + threadIdx.x) * dataPerThread.x;
	const int startJ = (blockIdx.y * blockDim.y + threadIdx.y) * dataPerThread.y;

	unsigned int rangeI = (startI + dataPerThread.x) < gridSize.x ? (startI + dataPerThread.x) : gridSize.x;
	unsigned int rangeJ = (startJ + dataPerThread.y) < gridSize.y ? (startJ + dataPerThread.y) : gridSize.y;

	//setGridColors
	for(unsigned int i = startI; i < rangeI; ++i)
	{
		for(unsigned int j = startJ; j < rangeJ; ++j)
		{
			float amplifier = 100.f;
			float grayScale = abs(grid[i * gridSize.y + j].soundPressure) * amplifier;
			Vertex & v = vertexPointer[(i * gridSize.y + j)];
			v.color = { grayScale, 1.f, 1.f };
		}
	}
}

__global__ void updatePoint(glm::ivec2 gridSize, AcousticFDTD::SpacePoint * grid, glm::ivec2 point, float sigPoint)
{
	grid[point.x * gridSize.y + point.y].soundPressure += sigPoint;
}

void AcousticFDTD::draw()
{
	CudaSafeCall(hipGraphicsMapResources(1, &_cudaVboRes, 0));
	size_t size;
	CudaSafeCall(hipGraphicsResourceGetMappedPointer((void **)(&_vertexPointer), &size, _cudaVboRes));

	updateV<<<_cudaGridSize, _cudaBlockSize>>>(_dataPerThread, _gridSize, _grid[(int)!_bufferSwap],
												_grid[(int)_bufferSwap], _dtOverDx, _density);
	hipDeviceSynchronize();
	CudaCheckError();

	updateP<<<_cudaGridSize, _cudaBlockSize>>>(_dataPerThread, _gridSize, _grid[(int)!_bufferSwap],
												_grid[(int)_bufferSwap], _dtOverDx, _bulkModulus);
	hipDeviceSynchronize();
	CudaCheckError();

	mur2nd<<<_cudaGridSize, _cudaBlockSize>>>(_dataPerThread, _gridSize, _grid[(int)_bufferSwap],
												_grid[(int)_bufferSwap], _murX[0], _murX[1], _murY[0], _murY[1],
												_dt, _dx, _density, _bulkModulus);
	hipDeviceSynchronize();
	CudaCheckError();

	//copy previous values
	mur2ndCopy<<<_cudaGridSize, _cudaBlockSize>>>(_dataPerThread, _gridSize, _grid[(int)_bufferSwap],
	                                            _murX[0], _murX[1], _murY[0], _murY[1]);
	hipDeviceSynchronize();
	CudaCheckError();

	// Initial Waveform from a Point Source (1 pulse of sinusoidal wave with Hann window)
	if( _nPoint < (1.0/_freq)/_dt ){
		_sigPoint = (1.0-cos((2.0*M_PI*_freq*_nPoint*_dt)))/2.0 * sin((2.0*M_PI*_freq*_nPoint*_dt));
		updatePoint<<<1, 1>>>(_gridSize, _grid[(int)_bufferSwap], _pointSource, _sigPoint);
		hipDeviceSynchronize();
		CudaCheckError();
	}

	updateColors<<<_cudaGridSize, _cudaBlockSize>>>(_dataPerThread, _gridSize, _grid[(int)_bufferSwap], _vertexPointer);
	hipDeviceSynchronize();
	CudaCheckError();

	CudaSafeCall(hipGraphicsUnmapResources(1, &_cudaVboRes, 0));

	++_nPoint;

	if(_nPoint > _randomPointSourceInterval)
	{
		randomPointSource();
	}

	_bufferSwap = !_bufferSwap;
}

void AcousticFDTD::randomPointSource()
{
	_pointSource.x = rand() % _gridSize.x;
	_pointSource.y = rand() % _gridSize.y;
	_nPoint = 0;
}
