#include "hip/hip_runtime.h"
#include "AcousticFDTD.h"

#define _USE_MATH_DEFINES
#include <math.h>

#include <CudaHelper.h>

AcousticFDTD::AcousticFDTD(glm::ivec2 & gridSize, GLuint * vbo)
{
	_gridSize = gridSize;
	_dataPerThread = glm::vec2(1, 1);

	//CudaSafeCall(hipGraphicsGLRegisterBuffer(&_cudaVboRes, *vbo, cudaGraphicsMapFlagsNone));

	//_cudaBlockSize = dim3(gridSize.x / _dataPerThread.x, gridSize.y / _dataPerThread.y);
	_cudaBlockSize = dim3(32, 32);
	const int bx = (gridSize.x + _cudaBlockSize.x - 1) / _cudaBlockSize.x;
	const int by = (gridSize.y + _cudaBlockSize.y - 1) / _cudaBlockSize.y;
	_cudaGridSize = dim3(bx, by);

	for(unsigned int i = 0; i < 2; ++i)
	{
		hipMalloc((void **)&_grid[i], (gridSize.x + 1) * (gridSize.y + 1) * sizeof(SpacePoint));
		hipMemset(_grid[i], 0, (gridSize.x + 1) * (gridSize.y + 1) * sizeof(SpacePoint));
		hipMalloc((void **)&_murX[i], gridSize.y * 4 * sizeof(float));
		hipMemset(_murX[i], 0, gridSize.y * 4 * sizeof(float));
		hipMalloc((void **)&_murY[i], gridSize.x * 4 * sizeof(float));
		hipMemset(_murY[i], 0, gridSize.x * 4 * sizeof(float));
	}
}

AcousticFDTD::~AcousticFDTD()
{
	CudaSafeCall(hipGraphicsUnregisterResource(_cudaVboRes));

	hipFree(_grid[0]); hipFree(_grid[1]);
	hipFree(_murX[0]); hipFree(_murX[1]);
	hipFree(_murY[0]); hipFree(_murY[1]);
}

__global__ void updateV(glm::ivec2 dataPerThread, glm::ivec2 gridSize, AcousticFDTD::SpacePoint * inGrid,
		AcousticFDTD::SpacePoint * outGrid, float dtOverDx, float density)
{
	const int startI = (blockIdx.x * blockDim.x + threadIdx.x) * dataPerThread.x;
	const int startJ = (blockIdx.y * blockDim.y + threadIdx.y) * dataPerThread.y;

	unsigned int rangeI = (startI + dataPerThread.x) < gridSize.x ? (startI + dataPerThread.x) : gridSize.x;
	unsigned int rangeJ = (startJ + dataPerThread.y) < gridSize.y ? (startJ + dataPerThread.y) : gridSize.y;

	for(unsigned int i = startI == 0 ? 1 : startI; i < rangeI; ++i)
	{
		for(unsigned int j = startJ; j < rangeJ; ++j)
		{
			outGrid[((i * gridSize.y) + j)].vx += -dtOverDx / density * (inGrid[(i * gridSize.y) + j].soundPressure - inGrid[((i - 1) * gridSize.y) + j].soundPressure);
		}
	}

	for(unsigned int i = startI; i < rangeI; ++i)
	{
		for(unsigned int j = startJ == 0 ? 1 : startJ; j < rangeJ; ++j)
		{
			outGrid[((i * gridSize.y) + j)].vy += -dtOverDx / density * (inGrid[(i * gridSize.y) + j].soundPressure - inGrid[i * gridSize.y + (j - 1)].soundPressure);
		}
	}
}

__global__ void updateP(glm::ivec2 dataPerThread, glm::ivec2 gridSize, AcousticFDTD::SpacePoint * inGrid,
		AcousticFDTD::SpacePoint * outGrid, float dtOverDx, float bulkModulus)
{
	const int startI = (blockIdx.x * blockDim.x + threadIdx.x) * dataPerThread.x;
	const int startJ = (blockIdx.y * blockDim.y + threadIdx.y) * dataPerThread.y;

	unsigned int rangeI = (startI + dataPerThread.x) < gridSize.x ? (startI + dataPerThread.x) : gridSize.x;
	unsigned int rangeJ = (startJ + dataPerThread.y) < gridSize.y ? (startJ + dataPerThread.y) : gridSize.y;

	for(unsigned int i = startI; i < rangeI; ++i)
	{
		for(unsigned int j = startJ; j < rangeJ; ++j)
		{
			outGrid[((i * gridSize.y) + j)].soundPressure += -(bulkModulus * dtOverDx) * ((inGrid[((i + 1) * gridSize.y) + j].vx - inGrid[(i * gridSize.y) + j].vx) + (inGrid[(i * gridSize.y) + (j + 1)].vy - inGrid[(i * gridSize.y) + j].vy));
		}
	}
}

__global__ void mur2nd(glm::ivec2 dataPerThread, glm::ivec2 gridSize, AcousticFDTD::SpacePoint * inGrid, AcousticFDTD::SpacePoint * outGrid,
		float * murX[2], float * murY[2], float dt, float dx, float density, float bulkModulus)
{
	const int startI = (blockIdx.x * blockDim.x + threadIdx.x) * dataPerThread.x;
	const int startJ = (blockIdx.y * blockDim.y + threadIdx.y) * dataPerThread.y;

	unsigned int rangeI = (startI + dataPerThread.x) < (gridSize.x - 2) ? (startI + dataPerThread.x) : (gridSize.x - 2);
	unsigned int rangeJ = (startJ + dataPerThread.y) < (gridSize.y - 2) ? (startJ + dataPerThread.y) : (gridSize.y - 2);

	float v = sqrt(bulkModulus/density);	// Wave velocity
	int i,j;
	/////
	outGrid[i * gridSize.y].soundPressure = murY[0][0];

//	for(i = startI < 2 ? 2 : startI; i < rangeI; ++i){
//		outGrid[(i * gridSize.y)].soundPressure = - murY[1][(i * 4 + 1)]
//		                                         + (v*dt-dx)/(v*dt+dx) * ( inGrid[(i * gridSize.y + 1)].soundPressure + murY[1][(i * 4)] )
//		                                         + (2.0*dx)/(v*dt+dx) * ( murY[0][i * 4] + murY[0][i * 4 + 1] )
//		                                         + (dx*v*v*dt*dt)/(2.0*dx*dx*(v*dt+dx))
//		                                           * ( murY[0][(i+1) * 4] - 2.0 * murY[0][i * 4]
//		                                               + murY[0][(i-1) * 4] + murY[0][(i+1) * 4 + 1]
//		                                               - 2.0 * murY[0][i* 4 + 1] + murY[0][(i-1) * 4 + 1] );
//		outGrid[(i * gridSize.y) + (gridSize.y - 1)].soundPressure = - murY[1][(i * 4) + 2]
//		                                                             + (v*dt-dx)/(v*dt+dx) * ( inGrid[(i * gridSize.y) + gridSize.y-2].soundPressure + murY[1][i * 4 + 3] )
//		                                                             + (2.0*dx)/(v*dt+dx) * ( murY[0][i * 4 + 3] + murY[0][i * 4 + 2] )
//		                                                             + (dx*v*v*dt*dt)/(2.0*dx*dx*(v*dt+dx))
//		                                                               * ( murY[0][(i+1) * 4 + 3] - 2.0 * murY[0][i * 4 + 3]
//		                                                                   + murY[0][(i-1) * 4 + 3] + murY[0][(i+1) * 4 + 2]
//		                                                                   - 2.0 * murY[0][i * 4 + 2] + murY[0][(i-1) * 4 + 2] );
//	}
//	for(j = startJ < 2 ? 2 : startJ; j < rangeJ; ++j){
//		outGrid[j].soundPressure = - murX[1][1 + j * 4]
//		                         + (v*dt-dx)/(v*dt+dx) * ( inGrid[1 * gridSize.y + j].soundPressure + murX[1][0 + j * 4] )
//		                         + (2.0*dx)/(v*dt+dx) * ( murX[0][0 + j * 4] + murX[0][1 + j * 4] )
//		                         + (dx*v*v*dt*dt)/(2.0*dx*dx*(v*dt+dx))
//		                           * ( murX[0][0 + (j+1) * 4] - 2.0 * murX[0][0 + j * 4]
//		                               + murX[0][0 + (j-1) * 4] + murX[0][1 + (j+1) * 4]
//		                               - 2.0 * murX[0][1 + j * 4] + murX[0][1 + (j-1) * 4] );
//		outGrid[(gridSize.x-1) * gridSize.y + j].soundPressure = - murX[1][2 + j * 4]
//		                                                         + (v*dt-dx)/(v*dt+dx) * ( inGrid[(gridSize.x-2) * gridSize.y + j].soundPressure + murX[1][3 + j * 4] )
//		                                                         + (2.0*dx)/(v*dt+dx) * ( murX[0][3 + j * 4] + murX[0][2 + j * 4] )
//		                                                         + (dx*v*v*dt*dt)/(2.0*dx*dx*(v*dt+dx))
//		                                                           * ( murX[0][3 + (j+1) * 4] - 2.0 * murX[0][3 + j * 4]
//		                                                               + murX[0][3 + (j-1) * 4] + murX[0][2 + (j+1) * 4]
//		                                                               - 2.0 * murX[0][2 + j * 4] + murX[0][2 + (j-1) * 4] );
//	}

	// corners are computed by first thread:
	// Mur's 1st Order Absorption for 4 corners
//	if(startI == 0)
//	{
//		i = 1;
//		outGrid[i * gridSize.y].soundPressure = murY[0][i * 4 + 1] + (v*dt-dx)/(v*dt+dx) * (inGrid[i * gridSize.y + 1].soundPressure - murY[0][i * 4]);
//		outGrid[i * gridSize.y + gridSize.y-1].soundPressure = murY[0][i * 4 + 2] + (v*dt-dx)/(v*dt+dx) * (inGrid[i * gridSize.y + gridSize.y-2].soundPressure - murY[0][i * 4 + 3]);
//		i = gridSize.x-2;
//		outGrid[i * gridSize.y].soundPressure = murY[0][i * 4 + 1] + (v*dt-dx)/(v*dt+dx) * (inGrid[i* gridSize.y + 1].soundPressure - murY[0][i * 4]);
//		outGrid[i * gridSize.y + gridSize.y-1].soundPressure = murY[0][i * 4 + 2] + (v*dt-dx)/(v*dt+dx) * (inGrid[i * gridSize.y + gridSize.y-2].soundPressure - murY[0][i * 4 + 3]);
//	}
//	if(startJ == 0)
//	{
//		j = 1;
//		outGrid[0 * gridSize.y + j].soundPressure = murX[0][1 + j * 4] + (v*dt-dx)/(v*dt+dx) * (inGrid[1 * gridSize.y + j].soundPressure - murX[0][0 + j * 4]);
//		outGrid[(gridSize.x-1) * gridSize.y + j].soundPressure = murX[0][2 + j * 4] + (v*dt-dx)/(v*dt+dx) * (inGrid[(gridSize.x-2) * gridSize.y + j].soundPressure - murX[0][3 + j * 4]);
//		j = gridSize.y - 2;
//		outGrid[0 * gridSize.y + j].soundPressure = murX[0][1 + j * 4] + (v*dt-dx)/(v*dt+dx) * (inGrid[1 * gridSize.y + j].soundPressure - murX[0][0 + j * 4]);
//		outGrid[(gridSize.x-1) * gridSize.y + j].soundPressure = murX[0][2+ j * 4] + (v*dt-dx)/(v*dt+dx) * (inGrid[(gridSize.x-2) * gridSize.y + j].soundPressure - murX[0][3 + j * 4]);
//	}
}

__global__ void mur2ndCopy(glm::ivec2 dataPerThread, glm::ivec2 gridSize, AcousticFDTD::SpacePoint * grid, float * murX[2], float * murY[2])
{
	const int startI = (blockIdx.x * blockDim.x + threadIdx.x) * dataPerThread.x;
	const int startJ = (blockIdx.y * blockDim.y + threadIdx.y) * dataPerThread.y;

	unsigned int rangeI = (startI + dataPerThread.x) < gridSize.x ? (startI + dataPerThread.x) : gridSize.x;
	unsigned int rangeJ = (startJ + dataPerThread.y) < gridSize.y ? (startJ + dataPerThread.y) : gridSize.y;

	for(unsigned int i = startI; i < rangeI; ++i)
	{
		/* Copy 1st Old Values to 2nd Old Values*/
		for(unsigned int j = 0; j < 4; ++j)
		{
			murY[1][i * 4 + j] = murY[0][i * 4 + j];
		}

		/* Copy Present Values */
		murY[0][i * 4 + 0] = grid[i * gridSize.y + 0].soundPressure;
		murY[0][i * 4 + 1] = grid[i * gridSize.y + 1].soundPressure;
		murY[0][i * 4 + 2] = grid[i * gridSize.y + gridSize.y-2].soundPressure;
		murY[0][i * 4 + 3] = grid[i * gridSize.y + gridSize.y-1].soundPressure;
	}
	for(unsigned int j = startJ; j < rangeJ; ++j){
		/* Copy 1st Old Values to 2nd Old Values*/
		for(unsigned int i = 0; i < 4; ++i)
		{
			murX[1][i + j * 4] = murX[0][i + j * 4];
		}

		/* Copy Present Values */
		murX[0][0 + j * 4] = grid[0 * gridSize.y + j].soundPressure;
		murX[0][1 + j * 4] = grid[1 * gridSize.y + j].soundPressure;
		murX[0][2 + j * 4] = grid[(gridSize.x-2) * gridSize.y + j].soundPressure;
		murX[0][3 + j * 4] = grid[(gridSize.x-1) * gridSize.y + j].soundPressure;
	}
}

__global__ void updateColors(glm::ivec2 dataPerThread, glm::ivec2 gridSize, AcousticFDTD::SpacePoint * grid, Vertex * vertexPointer)
{
	const int startI = (blockIdx.x * blockDim.x + threadIdx.x) * dataPerThread.x;
	const int startJ = (blockIdx.y * blockDim.y + threadIdx.y) * dataPerThread.y;

	unsigned int rangeI = (startI + dataPerThread.x) < gridSize.x ? (startI + dataPerThread.x) : gridSize.x;
	unsigned int rangeJ = (startJ + dataPerThread.y) < gridSize.y ? (startJ + dataPerThread.y) : gridSize.y;

	//setGridColors
	for(unsigned int i = startI; i < rangeI; ++i)
	{
		for(unsigned int j = startJ; j < rangeJ; ++j)
		{
			float amplifier = 100.f;
			float grayScale = abs(grid[i * gridSize.y + j].soundPressure) * amplifier;
			Vertex & v = vertexPointer[(i * gridSize.y + j)];
			//v.color = { 0.9, 1.0, 0.0 };
			v.color = { grayScale, grayScale, grayScale };
			// 2 * index + 1 because vbo consists of 2 vec3 (position and color)
			//vertexPointer[2 * (i * gridSize.y + j) + 1] = { grayScale, grayScale, grayScale };
			//vertexPointer[1 * (i * gridSize.y + j) + 1] = {0.5, 1.0, 0.0};
		}
	}
}

__global__ void updatePoint(glm::ivec2 gridSize, AcousticFDTD::SpacePoint * grid, glm::ivec2 point, float sigPoint)
{
	grid[point.x * gridSize.y + point.y].soundPressure += sigPoint;
}

void AcousticFDTD::draw()
{
//	CudaSafeCall(hipGraphicsMapResources(1, &_cudaVboRes, 0));
//	size_t size;
//	CudaSafeCall(hipGraphicsResourceGetMappedPointer((void **)(&_vertexPointer), &size, _cudaVboRes));

	updateV<<<_cudaGridSize, _cudaBlockSize>>>(_dataPerThread, _gridSize, _grid[(int)!_bufferSwap],
												_grid[(int)_bufferSwap], _dtOverDx, _density);
	hipDeviceSynchronize();
	CudaCheckError();

	updateP<<<_cudaGridSize, _cudaBlockSize>>>(_dataPerThread, _gridSize, _grid[(int)!_bufferSwap],
												_grid[(int)_bufferSwap], _dtOverDx, _bulkModulus);
	hipDeviceSynchronize();
	CudaCheckError();

	mur2nd<<<_cudaGridSize, _cudaBlockSize>>>(_dataPerThread, _gridSize, _grid[(int)!_bufferSwap],
												_grid[(int)_bufferSwap], _murX, _murY, _dt, _dx, _density, _bulkModulus);
	hipDeviceSynchronize();
	CudaCheckError();

	//copy previous values
	//mur2ndCopy<<<_cudaGridSize, _cudaBlockSize>>>(_dataPerThread, _gridSize, _grid[(int)_bufferSwap],
	                                          //_murX, _murY);
	hipDeviceSynchronize();
	CudaCheckError();

	// Initial Waveform from a Point Source (1 pulse of sinusoidal wave with Hann window)
	if( _nPoint < (1.0/_freq)/_dt ){
		_sigPoint = (1.0-cos((2.0*M_PI*_freq*_nPoint*_dt)))/2.0 * sin((2.0*M_PI*_freq*_nPoint*_dt));
		updatePoint<<<1, 1>>>(_gridSize, _grid[(int)_bufferSwap], _pointSource, _sigPoint);
		hipDeviceSynchronize();
		CudaCheckError();
	}

	//updateColors<<<_cudaGridSize, _cudaBlockSize>>>(_dataPerThread, _gridSize, _grid[(int)_bufferSwap], _vertexPointer);
	//hipDeviceSynchronize();
	//CudaCheckError();

//	CudaSafeCall(hipGraphicsUnmapResources(1, &_cudaVboRes, 0));

	++_nPoint;

	if(_nPoint > _randomPointSourceInterval)
	{
		randomPointSource();
	}

	_bufferSwap = !_bufferSwap;
}

void AcousticFDTD::randomPointSource()
{
	_pointSource.x = rand() % _gridSize.x;
	_pointSource.y = rand() % _gridSize.y;
	_nPoint = 0;
}
